/*!
 * LayoutKernel.cu
 * Projekt 3DVisual
 */
 
#ifndef GPU_LAYOUTKERNEL
#define GPU_LAYOUTKERNEL 1

#include <stdio.h>

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float3 operator+(float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

//------------------------------------------------------------------------------
inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void layoutKernel( float3* positions )
{
    unsigned int ptclIdx = thIdx();

    // perform a euler step
	positions[ptclIdx] = positions[ptclIdx] + make_float3(0,0.5f,0);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------

//checkCudaError
void checkCudaError(const char* message) 
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "CUDA error. %s. %s.\n", message, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

extern "C" __host__
void computeLayout( unsigned int numBlocks, unsigned int numThreads, void* positions )
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    layoutKernel<<< blocks, threads >>>(reinterpret_cast<float3*>(positions));
//	checkCudaError("Kernel Execution Failed!");
}

#endif
