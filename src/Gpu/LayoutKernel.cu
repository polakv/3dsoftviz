#include "hip/hip_runtime.h"
/*!
 * LayoutKernel.cu
 * Projekt 3DVisual
 */
 
#ifndef GPU_LAYOUTKERNEL
#define GPU_LAYOUTKERNEL 1

#include <math.h>
#include <stdio.h>

#define NUM_REPULSE_THREADS 32
#define NUM_ATTRACT_THREADS 32

__constant__ float calmEdgeLength;
__constant__ float alpha = 0.005;
__constant__ float maxMovement = 30;

texture<float4, hipTextureType1D, hipReadModeElementType> texVertices;
texture<uint2, hipTextureType1D, hipReadModeElementType> texEdges;
texture<float4, hipTextureType1D, hipReadModeElementType> texForces;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

inline __device__ 
float4 operator*(float4 a, float b)
{
    return make_float4(a.x * b, a.y * b, a.z * b, a.w * b);
}

inline __device__ 
float3 vertexVertexRepulsion(float4 u, float4 v, float3 fv)
{
	float3 r;  
	// r_uv  [3 FLOPS]  
	r.x = v.x - u.x;  
	r.y = v.y - u.y;  
	r.z = v.z - u.z;  
	// distSqr = dot(r_ij, r_ij) [5 FLOPS] 
	float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
	// distSqr = distSqr + small number [1 FLOP] -  to avoid division by zero
	distSqr = distSqr + 0.01f;
	// repForce =-k^2/distSqr  [3 FLOPS (2 mul, 1 inv)]  
	float repForce = -1.0f * calmEdgeLength * calmEdgeLength / distSqr;   
	// a_i =  a_i + s * r_ij [6 FLOPS]  
	fv.x += r.x * repForce;  
	fv.y += r.y * repForce;  
	fv.z += r.z * repForce;

	return fv;  
}

inline __device__ 
float3 tile_calculation(float4 vertexPosition, float3 fv)  
{   
	extern __shared__ float4 shPosition[];
	for (int i = 0; i < blockDim.x; i++) 
	{  
		fv = vertexVertexRepulsion(vertexPosition, shPosition[i], fv);  
	}  
	return fv;  
}

inline __device__ 
float3 edgeAttraction(float4 u, float4 v)
{
	float3 fv;  
	// r_uv  [3 FLOPS]  
	fv.x = v.x - u.x;  
	fv.y = v.y - u.y;  
	fv.z = v.z - u.z;   

	// dist [6 FLOPS (3 mul, 2 add, 1sqrt)]
	float dist = sqrtf(fv.x * fv.x + fv.y * fv.y + fv.z * fv.z);
	// attrForce = dist / calmEdgeLength [1 FLOP] 
	float attrForce = dist / calmEdgeLength;

	// fv =  fv * attrForce [3 FLOPS]  
	fv.x = fv.x * attrForce;  
	fv.y = fv.y * attrForce;  
	fv.z = fv.z * attrForce;

	return fv;  
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void repulseKernel( float4* vertices, float4* forceVectors, unsigned int numVertices )
{
    extern __shared__ float4 shPosition[];  

	int vertexId = blockIdx.x * blockDim.x + threadIdx.x;  
	float4 vertexPosition = vertices[vertexId];  
	float3 fv = {0.0f, 0.0f, 0.0f};
	for (int i = 0, tile = 0; i < numVertices; i += NUM_REPULSE_THREADS, tile++) 
	{  
		int idx = tile * blockDim.x + threadIdx.x;  
		shPosition[threadIdx.x] = vertices[idx];  
		__syncthreads();  
		fv = tile_calculation(vertexPosition, fv);  
		__syncthreads();  
	}  
	// save the result in global memory for the integration step.  
	forceVectors[vertexId] = make_float4(fv.x, fv.y, fv.z, 0.0f);
}

__global__
void attractKernel( float4* forceVectors )
{
	//TODO: what happend if edge idx is out of range
    unsigned int edgeIdx = blockDim.x * blockIdx.x + threadIdx.x;
	uint2 edge = tex1Dfetch(texEdges, edgeIdx);

	float4 u = tex1Dfetch(texVertices, edge.x);
	float4 v = tex1Dfetch(texVertices, edge.y);

	float3 fv = edgeAttraction(u, v);

	atomicAdd(&(forceVectors[edge.x].x), fv.x);
	atomicAdd(&(forceVectors[edge.x].y), fv.y);
	atomicAdd(&(forceVectors[edge.x].z), fv.z);

	atomicAdd(&(forceVectors[edge.y].x), 0 - fv.x);
	atomicAdd(&(forceVectors[edge.y].y), 0 - fv.y);
	atomicAdd(&(forceVectors[edge.y].z), 0 - fv.z);

}

__global__
void applyKernel( float4* vertices )
{
	unsigned int vertexIdx = blockDim.x * blockIdx.x + threadIdx.x;
	float4 force = tex1Dfetch(texForces, vertexIdx) * alpha;
	float length = sqrtf((force.x * force.x) + (force.y * force.y) + (force.z * force.z));

	if(length > maxMovement)
	{
		force.x = force.x / length;
		force.y = force.y / length;
		force.z = force.z / length;
		force = force * 5;
	}

	vertices[vertexIdx] = vertices[vertexIdx] + force;
}

__global__
void explosionKernel( float4* nodes )
{
    unsigned int ptclIdx = blockDim.x * blockIdx.x + threadIdx.x;

	float4 node =  nodes[ptclIdx];
	float length = sqrtf(powf(node.x,2) + powf(node.y,2) + powf(node.z,2));
  
	nodes[ptclIdx] = node + make_float4(node.x / length, node.y / length, node.z / length, 0.0f);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------

void checkCudaError(const char* message);
float computeCalm(unsigned int numVertices, float sizeFactor);

extern "C" __host__
void initKernelConstants(unsigned int numVertices, float sizeFactor)
{
	float calmEdgeLength = computeCalm(numVertices, sizeFactor);
	hipMemcpyToSymbol(HIP_SYMBOL("calmEdgeLength"), &calmEdgeLength, sizeof(float));
}

float computeCalm(unsigned int numVertices, float sizeFactor)
{
	float R = 30;
	float PI = acos((float) - 1);

	return sizeFactor * pow((4 * PI * R * R * R)/(numVertices * 3), (float) 1/3);
}

extern "C" __host__
void computeLayout(void* vertexBuffer, unsigned int vertexBufferSize,
				   void* edgeBuffer, unsigned int edgeBufferSize)
{
	//init
	unsigned int numBlocks, numThreads;
	dim3 blocks, threads;

	float4* forceVectorBuffer;
	size_t forceVectorSize = vertexBufferSize * sizeof(float4);
	hipMalloc(&forceVectorBuffer, forceVectorSize);


	//repulse kernel
    numThreads = NUM_REPULSE_THREADS;
	numBlocks = (unsigned int) ceil((float) vertexBufferSize / (float) numThreads);
	unsigned int sharedMemSize = numThreads * sizeof(float4);

	//printf("Repulse - blocks: %d threads: %d\n", numBlocks, numThreads);
	
	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

    repulseKernel<<< blocks, threads, sharedMemSize >>>(reinterpret_cast<float4*>(vertexBuffer), forceVectorBuffer, vertexBufferSize);

	//attract kernel
	numThreads = NUM_ATTRACT_THREADS;
	numBlocks = (unsigned int) ceil((float) edgeBufferSize / (float) (numThreads * 2));

	//printf("Attract - blocks: %d threads: %d\n", numBlocks, numThreads);

	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

	hipBindTexture(0, texVertices, reinterpret_cast<float4*>(vertexBuffer), vertexBufferSize * sizeof(float4));
	hipBindTexture(0, texEdges, reinterpret_cast<uint2*>(edgeBuffer), (edgeBufferSize / 2) * sizeof(uint2));

	attractKernel<<< blocks, threads >>>(forceVectorBuffer);

	hipUnbindTexture(texVertices);
	hipUnbindTexture(texEdges);

	//apply forces
	numThreads = NUM_REPULSE_THREADS;
	numBlocks = (unsigned int) ceil((float) vertexBufferSize / (float) numThreads);

	//printf("Apply - blocks: %d threads: %d\n", numBlocks, numThreads);
	
	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

	hipBindTexture(0, texForces, reinterpret_cast<float4*>(forceVectorBuffer), vertexBufferSize * sizeof(float4));

    applyKernel<<< blocks, threads >>>(reinterpret_cast<float4*>(vertexBuffer));

	hipUnbindTexture(texForces);

	checkCudaError("Kernel Execution Failed!");
}

extern "C" __host__
void createExplosion(void* nodes, unsigned int numNodes)
{
	unsigned int numThreads = 128;
	unsigned int numBlocks = (unsigned int) ceil((float) numNodes / (float) numThreads);
	
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    explosionKernel<<< blocks, threads >>>(reinterpret_cast<float4*>(nodes));
}

void checkCudaError(const char* message) 
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "CUDA error. %s. %s.\n", message, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

#endif
