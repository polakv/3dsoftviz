#include "hip/hip_runtime.h"
/*!
 * LayoutKernel.cu
 * Projekt 3DVisual
 */
 
#ifndef GPU_LAYOUTKERNEL
#define GPU_LAYOUTKERNEL 1

#include <math.h>
#include <stdio.h>

texture<uint2, hipTextureType1D, hipReadModeElementType> texEdgeIndexes;
texture<uint1, hipTextureType1D, hipReadModeElementType> texEdgeValues;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
inline __device__ 
float3 operator+(float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

inline __device__ 
float vectorLength(float3 position)
{
    return sqrtf(powf(position.x,2) + powf(position.y,2) + powf(position.z,2));
}


inline __device__
unsigned int thIdx()
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int width = gridDim.x * blockDim.x;

    return y*width + x;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__
void layoutKernel( float3* nodes )
{
    unsigned int ptclIdx = thIdx();

	uint2 edgeIndex = tex1Dfetch(texEdgeIndexes, ptclIdx);
	uint1 edgeValue = tex1Dfetch(texEdgeValues, edgeIndex.x);

	float3 node =  nodes[ptclIdx];
	float length = vectorLength(node);
  
	nodes[ptclIdx] = node + make_float3(node.x / length, node.y / length, node.z / length);
}

__global__
void explosionKernel( float3* nodes )
{
    unsigned int ptclIdx = thIdx();

	float3 node =  nodes[ptclIdx];
	float length = vectorLength(node);
  
	nodes[ptclIdx] = node + make_float3(node.x / length, node.y / length, node.z / length);
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------

void checkCudaError(const char* message) 
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "CUDA error. %s. %s.\n", message, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

extern "C" __host__
void computeLayout( unsigned int numBlocks, unsigned int numThreads, void* nodes, void* edgeIndexes, unsigned int edgeIndexesSize, void* edgeValues, unsigned int edgeValuesSize)
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

	hipBindTexture(0, texEdgeIndexes, reinterpret_cast<uint2*>(edgeIndexes), (edgeIndexesSize / 2) * sizeof(uint2));
	hipBindTexture(0, texEdgeValues, reinterpret_cast<uint1*>(edgeValues), edgeValuesSize * sizeof(uint1));

    layoutKernel<<< blocks, threads >>>(reinterpret_cast<float3*>(nodes));

	hipUnbindTexture(texEdgeIndexes);
	hipUnbindTexture(texEdgeValues);

	//checkCudaError("Kernel Execution Failed!");
}

extern "C" __host__
void createExplosion( unsigned int numBlocks, unsigned int numThreads, void* nodes)
{
    dim3 blocks( numBlocks, 1, 1 );
    dim3 threads( numThreads, 1, 1 );

    explosionKernel<<< blocks, threads >>>(reinterpret_cast<float3*>(nodes));
}

#endif
