#include "hip/hip_runtime.h"
/*!
 * LayoutKernel.cu
 * Projekt 3DVisual
 */
 
#ifndef GPU_LAYOUTKERNEL
#define GPU_LAYOUTKERNEL 1

#include <stdio.h>
#include <math.h>

#define NUM_THREADS 256 

__constant__ float alpha;
__constant__ float minMovement;
__constant__ float maxMovement;
__constant__ float flexibility;
__constant__ float calmEdgeLength;

texture<float4, hipTextureType1D, hipReadModeElementType> texVertices;
texture<uint2, hipTextureType1D, hipReadModeElementType> texEdges;
texture<float4, hipTextureType1D, hipReadModeElementType> texForces;

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__device__ 
float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__device__ 
float4 operator*(float4 a, float b)
{
    return make_float4(a.x * b, a.y * b, a.z * b, a.w * b);
}

__device__
float3 vertexVertexRepulsion(float4 u, float4 v, float3 fv)
{
	float3 r;  
	// r_uv  [3 FLOPS]  
	r.x = v.x - u.x;  
	r.y = v.y - u.y;  
	r.z = v.z - u.z;
	// distSqr = dot(r_ij, r_ij) [5 FLOPS] 
	float distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
	// repForce = -k^2/distSqr  [4 FLOPS (2 mul, 1 add, 1 inv)]  
	float repForce = -1.0f * calmEdgeLength * calmEdgeLength / (distSqr + 0.0001f);  
	// ignore meta nodes
	unsigned int metaFlag = ((unsigned int) v.w) & 1;
	repForce *= (metaFlag ^ 1);
	// a_i =  a_i + s * r_ij [6 FLOPS]  
	fv.x += r.x * repForce;  
	fv.y += r.y * repForce;  
	fv.z += r.z * repForce;

	return fv;  
}

__device__
float3 tile_calculation(float4 vertexPosition, float3 fv, unsigned int arrayBound)  
{   
	extern __shared__ float4 shPosition[];
	#pragma unroll 32
	for (unsigned int i = 0; i < arrayBound; i++) 
	{  
		fv = vertexVertexRepulsion(vertexPosition, shPosition[i], fv); 
	}  
	return fv;  
}

__device__
float3 edgeAttraction(float4 u, float4 v)
{
	float3 fv;  
	// r_uv  [3 FLOPS]  
	fv.x = v.x - u.x;  
	fv.y = v.y - u.y;  
	fv.z = v.z - u.z;   

	// dist [6 FLOPS (3 mul, 2 add, 1sqrt)]
	float dist = sqrtf(fv.x * fv.x + fv.y * fv.y + fv.z * fv.z);
	// attrForce = dist / calmEdgeLength [1 FLOP] 
	float attrForce = dist / calmEdgeLength;

	// fv =  fv * attrForce [3 FLOPS]  
	fv.x = fv.x * attrForce;  
	fv.y = fv.y * attrForce;  
	fv.z = fv.z * attrForce;

	return fv;
}


//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS //////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------
__global__ 
void repulseKernel( float4* vertices, float4* forceVectors, unsigned int numVertices )
{
	int vertexIdx = blockIdx.x * blockDim.x + threadIdx.x; 
	if(vertexIdx >= numVertices)
	{
		return; 
	}

	extern __shared__ float4 shPosition[]; 
	float4 vertexPosition = vertices[vertexIdx];  
	float3 fv = {0.0f, 0.0f, 0.0f};
	for (int tile = 0; tile * blockDim.x < numVertices; tile++) 
	{
		shPosition[threadIdx.x] = tile * blockDim.x + threadIdx.x < numVertices ? vertices[tile * blockDim.x + threadIdx.x] : vertexPosition;
		__syncthreads();  
		unsigned int arrayBound = (numVertices - tile * blockDim.x) > blockDim.x ? blockDim.x : (numVertices - tile * blockDim.x);
		fv = tile_calculation(vertexPosition, fv, arrayBound);  
		__syncthreads();  
	}  
	// save the result in global memory for apply kernel  
	forceVectors[vertexIdx] = make_float4(fv.x, fv.y, fv.z, 0.0f);
}

__global__
void attractKernel( float4* forceVectors, unsigned int numEdges )
{
    unsigned int edgeIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if(edgeIdx >= numEdges)
	{
		return; 
	}

	uint2 edge = tex1Dfetch(texEdges, edgeIdx);

	float4 u = tex1Dfetch(texVertices, edge.x);
	float4 v = tex1Dfetch(texVertices, edge.y);

	float3 fv = edgeAttraction(u, v);

	atomicAdd(&(forceVectors[edge.x].x), fv.x);
	atomicAdd(&(forceVectors[edge.x].y), fv.y);
	atomicAdd(&(forceVectors[edge.x].z), fv.z);

	atomicAdd(&(forceVectors[edge.y].x), fv.x * -1);
	atomicAdd(&(forceVectors[edge.y].y), fv.y * -1);
	atomicAdd(&(forceVectors[edge.y].z), fv.z * -1);

}

__global__
void applyKernel( float4* vertices, unsigned int numVertices, float4* velocities )
{
	unsigned int vertexIdx = blockDim.x * blockIdx.x + threadIdx.x;
	if(vertexIdx >= numVertices)
	{
		return;
	}

	float4 force = tex1Dfetch(texForces, vertexIdx) * alpha;
	float length = sqrtf((force.x * force.x) + (force.y * force.y) + (force.z * force.z));

	force.x = force.x / length;
	force.y = force.y / length;
	force.z = force.z / length;
	
	float optimalLength = length < maxMovement ?  length : maxMovement;
	force = length > minMovement ? (force * optimalLength) + velocities[vertexIdx] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);

	unsigned int fixedFlag = ((unsigned int) vertices[vertexIdx].w) >> 1 & 1;
	force = force * (fixedFlag ^ 1);

	vertices[vertexIdx] = vertices[vertexIdx] + force;
	velocities[vertexIdx] = force * flexibility;
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////
// HOST FUNCTIONS ////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////
//------------------------------------------------------------------------------

float computeCalm(unsigned int numVertices, float sizeFactor)
{
	float R = 300;
	float PI = acos((float) - 1);

	return sizeFactor * pow((4 * PI * R * R * R)/(numVertices * 3), (float) 1/3);
}

extern "C" __host__
void initKernelConstants(float alphaValue, float minMovementValue, float maxMovementValue, float flexibilityValue, float sizeFactor, unsigned int numVertices)
{
	hipMemcpyToSymbol(HIP_SYMBOL(alpha), &alphaValue, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(minMovement), &minMovementValue, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(maxMovement), &maxMovementValue, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(flexibility), &flexibilityValue, sizeof(float));

	float calmEdgeLengthValue = computeCalm(numVertices, sizeFactor);
	hipMemcpyToSymbol(HIP_SYMBOL(calmEdgeLength), &calmEdgeLengthValue, sizeof(float));
}

void checkCudaError(const char* message) 
{
	hipError_t error = hipGetLastError();
	if (error != hipSuccess)
	{
		fprintf(stderr, "CUDA error. %s. %s.\n", message, hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

extern "C" __host__
void computeLayout(void* vertexBuffer, unsigned int vertexBufferSize, void* velocityBuffer,
				   void* edgeBuffer, unsigned int edgeBufferSize)
{
	//init
	unsigned int numBlocks, numThreads;
	dim3 blocks, threads;

	float4* forceVectorBuffer;
	size_t forceVectorSize = vertexBufferSize * sizeof(float4);
	hipMalloc(&forceVectorBuffer, forceVectorSize);

	//repulse kernel
    numThreads = NUM_THREADS;
	numBlocks = (unsigned int) ceil((float) vertexBufferSize / (float) numThreads);
	unsigned int sharedMemSize = numThreads * sizeof(float4);
	
	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

    repulseKernel<<< blocks, threads, sharedMemSize >>>(reinterpret_cast<float4*>(vertexBuffer), forceVectorBuffer, vertexBufferSize);

	//attract kernel
	unsigned int numEdges = edgeBufferSize / 2;
	numThreads = NUM_THREADS;
	numBlocks = (unsigned int) ceil((float) numEdges / (float) numThreads);

	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

	hipBindTexture(0, texVertices, reinterpret_cast<float4*>(vertexBuffer), vertexBufferSize * sizeof(float4));
	hipBindTexture(0, texEdges, reinterpret_cast<uint2*>(edgeBuffer), numEdges * sizeof(uint2));

	attractKernel<<< blocks, threads >>>(forceVectorBuffer, numEdges);

	hipUnbindTexture(texVertices);
	hipUnbindTexture(texEdges);

	//apply forces
	numThreads = NUM_THREADS;
	numBlocks = (unsigned int) ceil((float) vertexBufferSize / (float) numThreads);
	
	blocks = dim3( numBlocks, 1, 1 );
	threads = dim3( numThreads, 1, 1 );

	hipBindTexture(0, texForces, reinterpret_cast<float4*>(forceVectorBuffer), vertexBufferSize * sizeof(float4));

    applyKernel<<< blocks, threads >>>(reinterpret_cast<float4*>(vertexBuffer), vertexBufferSize, reinterpret_cast<float4*>(velocityBuffer));

	hipUnbindTexture(texForces);

	//cleanup
	hipFree(forceVectorBuffer);
	checkCudaError("Kernel Execution Failed!");
}

#endif
